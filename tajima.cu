#include "hip/hip_runtime.h"
#include "tajima.cuh"
#include "functions.cuh"
#include "prometheus.cuh"

tajima::tajima(string gene_List, string input_Folder, string ouput_Path, int cuda_ID, string intermediate_Path, int ploidy)
{
    cout << "Initiating CUDA powered Tajima's D calculator" << endl
         << endl;

    set_Values(gene_List, input_Folder, ouput_Path, cuda_ID, intermediate_Path, ploidy);

    // this->gene_List = gene_List;
    // cout << "Gene list file path\t: " << gene_List << endl
    //      << endl;
    // this->input_Folder = input_Folder;
    // this->ouput_Path = ouput_Path;
    // this->intermediate_Path = intermediate_Path;
    // this->ploidy = ploidy;

    // hipSetDevice(cuda_ID);
    // cout << "Properties of selected CUDA GPU:" << endl;
    // hipDeviceProp_t prop;
    // hipGetDeviceProperties(&prop, cuda_ID);
    // cout << "GPU number\t: " << cuda_ID << endl;
    // cout << "GPU name\t: " << prop.name << endl;
    // size_t l_free = 0;
    // size_t l_Total = 0;
    // hipError_t error_id = hipMemGetInfo(&l_free, &l_Total);
    // cout << "GPU memory (GB)\t: " << l_Total / (1000 * 1000 * 1000) << endl;
    // cout << "GPU number of multiprocessor(s)\t: " << prop.multiProcessorCount << endl;
    // cout << "GPU block(s) per multiprocessor\t: " << prop.maxBlocksPerMultiProcessor << endl;
    // this->tot_Blocks = prop.maxBlocksPerMultiProcessor;
    // this->tot_ThreadsperBlock = prop.maxThreadsPerBlock;
    // cout << "GPU thread(s) per block\t: " << tot_ThreadsperBlock << endl
    //      << endl;
}

tajima::tajima(string gene_List, string input_Folder, string ouput_Path, int cuda_ID, string intermediate_Path, int ploidy, string prometheus_Activate, string Multi_read, int number_of_genes, int CPU_cores, int SNPs_per_Run)
{
    // PROMETHEUS Constructor gene file
    cout << "Initiating CUDA powered Tajima's D calculator on PROMETHEUS" << endl
         << endl;

    set_Values(gene_List, input_Folder, ouput_Path, cuda_ID, intermediate_Path, ploidy);

    // this->gene_List = gene_List;
    // cout << "Gene list file path\t: " << gene_List << endl
    //      << endl;
    // this->input_Folder = input_Folder;
    // this->ouput_Path = ouput_Path;
    // this->intermediate_Path = intermediate_Path;
    // this->ploidy = ploidy;

    // hipSetDevice(cuda_ID);
    // cout << "Properties of selected CUDA GPU:" << endl;
    // hipDeviceProp_t prop;
    // hipGetDeviceProperties(&prop, cuda_ID);
    // cout << "GPU number\t: " << cuda_ID << endl;
    // cout << "GPU name\t: " << prop.name << endl;
    // size_t l_free = 0;
    // size_t l_Total = 0;
    // hipError_t error_id = hipMemGetInfo(&l_free, &l_Total);
    // cout << "GPU memory (GB)\t: " << l_Total / (1000 * 1000 * 1000) << endl;
    // cout << "GPU number of multiprocessor(s)\t: " << prop.multiProcessorCount << endl;
    // cout << "GPU block(s) per multiprocessor\t: " << prop.maxBlocksPerMultiProcessor << endl;
    // this->tot_Blocks = prop.maxBlocksPerMultiProcessor;
    // this->tot_ThreadsperBlock = prop.maxThreadsPerBlock;
    // cout << "GPU thread(s) per block\t: " << tot_ThreadsperBlock << endl
    //      << endl;

    this->prometheus_Activate = "YES";
    this->CPU_cores = CPU_cores;
    this->SNPs_per_Run = SNPs_per_Run;
    transform(Multi_read.begin(), Multi_read.end(), Multi_read.begin(), ::toupper);
    this->Multi_read = Multi_read;
    this->number_of_genes = number_of_genes;
}

tajima::tajima(string calc_Mode, int window_Size, int step_Size, string input_Folder, string ouput_Path, int cuda_ID, int ploidy, string prometheus_Activate, string Multi_read, int number_of_genes, int CPU_cores, int SNPs_per_Run)
{
    // PROMETHEUS WINDOW MODE CONSTRUCTOR

    cout << "Initiating CUDA powered Tajima's D calculator on PROMETHEUS" << endl
         << endl;
    this->calc_Mode = "WINDOW";
    set_Values("", input_Folder, ouput_Path, cuda_ID, "", ploidy);

    this->window_Size = window_Size;
    this->step_Size = step_Size;

    this->prometheus_Activate = "YES";
    this->CPU_cores = CPU_cores;
    this->SNPs_per_Run = SNPs_per_Run;
    transform(Multi_read.begin(), Multi_read.end(), Multi_read.begin(), ::toupper);
    this->Multi_read = Multi_read;
    this->number_of_genes = number_of_genes;
}

void tajima::set_Values(string gene_List, string input_Folder, string ouput_Path, int cuda_ID, string intermediate_Path, int ploidy)
{
    if (this->calc_Mode == "WINDOW")
    {
        cout << "Calculation mode: WINDOW" << endl
             << endl;
    }
    else
    {
        cout << "Calculation mode: FILE" << endl;
        this->gene_List = gene_List;
        cout << "Gene list file path\t: " << gene_List << endl
             << endl;
    }

    this->input_Folder = input_Folder;
    this->ouput_Path = ouput_Path;
    this->intermediate_Path = intermediate_Path;
    this->ploidy = ploidy;

    hipSetDevice(cuda_ID);
    cout << "Properties of selected CUDA GPU:" << endl;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_ID);
    cout << "GPU number\t: " << cuda_ID << endl;
    cout << "GPU name\t: " << prop.name << endl;
    size_t l_free = 0;
    size_t l_Total = 0;
    hipError_t error_id = hipMemGetInfo(&l_free, &l_Total);
    cout << "GPU memory (GB)\t: " << l_Total / (1000 * 1000 * 1000) << endl;
    cout << "GPU number of multiprocessor(s)\t: " << prop.multiProcessorCount << endl;
    cout << "GPU block(s) per multiprocessor\t: " << prop.maxBlocksPerMultiProcessor << endl;
    this->tot_Blocks = prop.maxBlocksPerMultiProcessor;
    this->tot_ThreadsperBlock = prop.maxThreadsPerBlock;
    cout << "GPU thread(s) per block\t: " << tot_ThreadsperBlock << endl
         << endl;
}

void tajima::ingress()
{
    functions function = functions();
    vector<string> countries = get_Countries();
    cout << countries.size() << " population(s) were found: ";
    for (int count = 0; count < countries.size(); count++)
    {
        string folder = countries[count];
        cout << folder.substr(folder.find_last_of("/") + 1, folder.length());
        if (count < countries.size() - 1)
        {
            cout << ", ";
        }
    }
    cout << endl
         << endl;
    for (string country : countries)
    {
        cout << "Processing country\t: " << country.substr(country.find_last_of("/") + 1, country.length()) << endl
             << endl;
        // first: start_stop second: filename
        vector<pair<string, string>> folder_Index = function.index_Folder(country);

        // for (auto file : folder_Index)
        // {
        //     cout << file.first << "\t" << file.second << endl;
        // }

        cout << "Completed indexing folder\t: " << country << endl;
        string check_AF_country = country.substr(country.find_last_of("/") + 1, country.length()) + "_AF";
        cout << endl;

        int samples = function.getN_Split(folder_Index[0].second);
        cout << "Number of samples in " << country.substr(country.find_last_of("/") + 1, country.length()) << " population\t: " << samples << endl;
        int N = samples * ploidy;
        cout << "Number of sequences in " << country.substr(country.find_last_of("/") + 1, country.length()) << " population [ " << samples << " x " << ploidy << " ] (N)\t: " << N << endl;
        long int combinations = combos_N(N);
        cout << "Pairwise combinations\t: " << combinations << endl;
        cout << endl;

        float a1, e1, e2;
        calc_Pre(N, a1, e1, e2);

        if (this->calc_Mode != "FILE")
        {
            string output_File = ouput_Path + "/" +
                                 country.substr(country.find_last_of("/") + 1, country.length()) + "_" +
                                 to_string(window_Size) + "_" + to_string(step_Size) +
                                 ".td";

            prometheus pro_Tajima_Window = prometheus(output_File, window_Size, step_Size, folder_Index, Multi_read, tot_Blocks, tot_ThreadsperBlock, combinations, a1, e1, e2, N, CPU_cores, SNPs_per_Run, number_of_genes);
            pro_Tajima_Window.process_Window("T");
        }
        else
        {
            string output_File = ouput_Path + "/" +
                                 country.substr(country.find_last_of("/") + 1, country.length()) + "_" +
                                 filesystem::path(gene_List).stem().string() +
                                 ".td";
            string intermediate_File = intermediate_Path + "/" +
                                       country.substr(country.find_last_of("/") + 1, country.length()) + "_" +
                                       filesystem::path(gene_List).stem().string() +
                                       ".log_td";

            fstream gene_File;
            gene_File.open(gene_List, ios::in);
            cout << "Processing gene list:" << endl;
            cout << endl;
            cout << "Writing to file\t: " << output_File << endl;
            cout << endl;

            if (gene_File.is_open())
            {
                string gene_Combo;

                if (filesystem::exists(output_File) == 0)
                {
                    createFile(output_File, "Gene_name\tCoordinates\tPi\tS\tTajimas_D");
                    createFile(intermediate_File);
                }
                else
                {
                    fstream intermediate;
                    intermediate.open(intermediate_File, ios::in);
                    string get_finished;
                    while (getline(intermediate, get_finished))
                    {
                        getline(gene_File, gene_Combo);
                        if (gene_Combo != get_finished)
                        {
                            break;
                        }
                    }
                    intermediate.close();
                }
                // cout << "CHECK: " << gene_Combo << endl;

                fstream output;
                fstream intermediate;
                output.open(output_File, ios::app);
                intermediate.open(intermediate_File, ios::app);

                // ADD Prometheus HERE
                if (prometheus_Activate == "YES")
                {
                    string test = "T";
                    cout << "Initializing Prometheus:" << endl
                         << endl;
                    // cout << "Processing on " << this->CPU_cores << " CPU cores" << endl;
                    // cout << "Processing " << this->number_of_genes << " genes at a time" << endl;
                    // cout << "Processing " << this->SNPs_per_Run << " SNPs at a time" << endl;
                    // if (this->Multi_read == "YES")
                    // {
                    //     cout << "Multi read: Available" << endl;
                    // }
                    // else
                    // {
                    //     cout << "Multi read: Unavailable" << endl;
                    // }
                    // cout << endl;

                    prometheus pro_Tajima = prometheus(folder_Index, Multi_read, this->tot_Blocks, this->tot_ThreadsperBlock, combinations, a1, e1, e2, N, CPU_cores, SNPs_per_Run, number_of_genes);

                    vector<string> gene_Collect;

                    while (getline(gene_File, gene_Combo))
                    {
                        gene_Collect.push_back(gene_Combo);
                        if (gene_Collect.size() == number_of_genes)
                        {
                            cout << "Prometheus batch intialized" << endl;
                            cout << "From: " << gene_Collect[0] << endl;
                            cout << "To  : " << gene_Collect[gene_Collect.size() - 1] << endl
                                 << endl;
                            // launch prometheus
                            vector<string> write_Lines = pro_Tajima.collection_Engine(gene_Collect, test);
                            // print
                            cout << "System is writing Tajima's D results" << endl;
                            for (size_t i = 0; i < write_Lines.size(); i++)
                            {
                                output << write_Lines[i] << "\n";
                                intermediate << gene_Combo << "\n";
                            }
                            // clear prometheus
                            output.flush();
                            intermediate.flush();
                            pro_Tajima.erase();
                            gene_Collect.clear();
                            cout << endl;
                        }
                    }

                    if (gene_Collect.size() != 0)
                    {
                        // RUN PROMETHEUS for remaining
                        // launch prometheus
                        cout << "Prometheus batch intialized" << endl;
                        cout << "From: " << gene_Collect[0] << endl;
                        cout << "To  : " << gene_Collect[gene_Collect.size() - 1] << endl
                             << endl;

                        vector<string> write_Lines = pro_Tajima.collection_Engine(gene_Collect, test);
                        // print
                        cout << "System is writing Tajima's D results" << endl;
                        for (size_t i = 0; i < write_Lines.size(); i++)
                        {
                            if (write_Lines[i] != "")
                            {
                                output << write_Lines[i] << "\n";
                                intermediate << gene_Combo << "\n";
                            }
                        }
                        cout << endl;
                    }

                    output.flush();
                    intermediate.flush();
                    pro_Tajima.erase();
                    gene_Collect.clear();

                    // cout << endl;
                }
                else
                {
                    while (getline(gene_File, gene_Combo))
                    {

                        // cout << gene_Combo << endl;
                        vector<string> split_Data;
                        function.split(split_Data, gene_Combo, '\t');
                        string gene_Name = split_Data[0];
                        cout << "Gene name\t: " << gene_Name << endl;
                        vector<string> coordinates;
                        function.split(coordinates, split_Data[1], ':');
                        int start_Co = stoi(coordinates[1]);
                        int end_Co = stoi(coordinates[2]);
                        cout << "Coordinates\t: Chromosome: " << coordinates[0] << " Start: " << start_Co << " End: " << end_Co << endl;

                        float tot_pairwise_Differences = 0;
                        // int tot_pairwise_Differences_TEST = 0;
                        int segregating_Sites = 0;

                        vector<string> file_List;
                        cout << endl;
                        cout << "System is retrieving file(s)" << endl;
                        if (folder_Index.size() > 1)
                        {
                            file_List = function.compound_interpolationSearch(folder_Index, start_Co, end_Co);
                        }
                        else
                        {
                            file_List.push_back(folder_Index[0].second);
                        }
                        cout << "System has retrieved all file(s)" << endl;

                        cout << "System is collecting segregrating site(s)" << endl;
                        vector<string> collect_Segregrating_sites;
                        for (string files : file_List)
                        {
                            // cout << files << endl;
                            fstream file;
                            file.open(files, ios::in);
                            if (file.is_open())
                            {
                                string line;
                                getline(file, line); // skip first header line
                                while (getline(file, line))
                                {
                                    vector<string> positions;
                                    function.split_getPos_ONLY(positions, line, '\t');
                                    int pos = stoi(positions[1]);

                                    if (pos >= start_Co && pos <= end_Co)
                                    {
                                        collect_Segregrating_sites.push_back(line);
                                        //     //cout << pos << endl;
                                        //     string check_0 = country.substr(country.find_last_of("/") + 1, country.length()) + "_AF=0";
                                        //     string GO = "GO";
                                        //     vector<string> info;
                                        //     split(info, positions[7], ";");
                                        //     for (string AF_check : info)
                                        //     {
                                        //         if (AF_check == check_0)
                                        //         {
                                        //             // cout << pos << endl;
                                        //             GO = "NO";
                                        //             break;
                                        //         }
                                        //     }
                                        //     if (GO == "GO")
                                        //     {
                                        //         //string check_AF_country = country.substr(country.find_last_of("/") + 1, country.length()) + "_AF";
                                        //         float MAF = 0.0000;
                                        //         for (string AF_check : info)
                                        //         {
                                        //             vector<string> split_info;
                                        //             split(split_info, AF_check, "=");
                                        //             if (split_info[0] == check_AF_country)
                                        //             {
                                        //                 MAF = stof(split_info[1]);
                                        //                 if (MAF > 0.5)
                                        //                 {
                                        //                     MAF = 1 - MAF;
                                        //                 }
                                        //                 //cout << split_info[0] << "\t: " << MAF << endl;
                                        //                 break;
                                        //             }
                                        //         }
                                        //         tot_pairwise_Differences = tot_pairwise_Differences + (MAF * (1 - MAF) * pow(N, 2));
                                        //         //cout << "pairwise differences: \t" << (MAF * (1 - MAF) * pow(N, 2)) << endl;
                                        //         //int pairwise_Differences = calc_Pairwise(line, N);
                                        //         //cout << pairwise_Differences << endl;
                                        //         //tot_pairwise_Differences = tot_pairwise_Differences + pairwise_Differences;
                                        //         segregating_Sites = segregating_Sites + 1;
                                        //         //break;
                                        //     }
                                    }
                                    else if (pos > end_Co)
                                    {
                                        break;
                                    }
                                }
                                file.close();
                            }
                        }

                        function.process_Seg_sites_tajima(collect_Segregrating_sites, N, segregating_Sites, tot_pairwise_Differences, this->tot_Blocks, this->tot_ThreadsperBlock);

                        cout << endl;
                        // cout << "totDif " << tot_pairwise_Differences << endl;
                        float pi = 0;
                        // float pi_Test = 0;
                        float D = 0;
                        string Tajima_D;
                        cout << "Total segregating sites (S)\t: " << segregating_Sites << endl;
                        if (segregating_Sites != 0)
                        {
                            // pi_Test = (float)tot_pairwise_Differences_TEST / combinations;
                            pi = (float)tot_pairwise_Differences / combinations;
                            // cout << "TEST Average pairwise polymorphisms (pi)\t: " << pi_Test << endl;
                            cout << "Average pairwise polymorphisms (pi)\t: " << pi << endl;
                            D = (float)(pi - (segregating_Sites / a1)) / sqrt(((e1 * segregating_Sites) + (e2 * segregating_Sites * (segregating_Sites - 1))));
                            cout << endl;
                            cout << "Tajima's D\t: " << D << endl;
                            Tajima_D = to_string(D);
                        }
                        else
                        {
                            cout << endl;
                            cout << "Tajima's D\t: "
                                 << "Not Available" << endl;
                            Tajima_D = "NA";
                        }

                        cout << endl;

                        //"Gene_name\tCoordinates\tPi\tS\tTajimas_D"
                        output << gene_Name << "\t"
                               << coordinates[0] << ":" << to_string(start_Co) << ":" << to_string(end_Co)
                               << "\t" << to_string(pi)
                               << "\t" << to_string(segregating_Sites)

                               << "\t" << Tajima_D << "\n";

                        intermediate << gene_Combo << "\n";
                        output.flush();
                        intermediate.flush();
                    }
                }
                output.close();
                intermediate.close();
                gene_File.close();
            }
        }
    }
}

void tajima::createFile(string path)
{
    fstream file;
    file.open(path, ios::out);
    file.close();
}

void tajima::createFile(string path, string text)
{
    fstream file;
    file.open(path, ios::out);
    file << text;
    file << "\n";
    file.close();
}

__global__ void pairwise_Cuda(int N, int *SNP, int *differences)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < N)
    {
        int tot = 0;
        for (int i = tid + 1; i < N; i++)
        {
            // printf("snp1 is %d and snp %d is %d \n",SNP[tid],i,SNP[i]);
            if (SNP[tid] != SNP[i])
            {
                tot = tot + 1;
            }
        }
        // printf("tid is %d and diff %d \n",tid,tot);
        differences[tid] = tot;
        tid += blockDim.x * gridDim.x;
    }
}

int tajima::calc_Pairwise(string &line, int N)
{
    int pairwise_Differences = 0;

    // int *line_temp = (int *)malloc(N * sizeof(int));
    int *line_temp = new int[N];
    split_Convert(line_temp, line, "\t");
    // for (int i = 0; i < N; i++)
    // {
    //     cout << i << "\t" << line_temp[i] << endl;
    // }

    int *cuda_line_Data;
    hipMallocManaged(&cuda_line_Data, N * sizeof(int));

    int *differences, *cuda_Differences;
    hipMallocManaged(&cuda_Differences, N * sizeof(int));
    differences = (int *)malloc(N * sizeof(int));

    hipMemcpy(cuda_line_Data, line_temp, (N * sizeof(int)), hipMemcpyHostToDevice);

    pairwise_Cuda<<<tot_Blocks, tot_ThreadsperBlock>>>(N, cuda_line_Data, cuda_Differences);
    hipDeviceSynchronize();

    hipMemcpy(differences, cuda_Differences, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(cuda_line_Data);
    hipFree(cuda_Differences);

    for (int i = 0; i < N; i++)
    {
        pairwise_Differences = pairwise_Differences + differences[i];
    }

    // cout << "pairwise: " << pairwise_Differences << endl;

    free(differences);
    free(line_temp);

    return pairwise_Differences;
}

void tajima::split_getPos(vector<string> &line_Data, string line, string delim)
{
    vector<string>().swap(line_Data);
    char *convert;
    string capture(line);
    convert = &capture[0];
    // cout<<convert;

    char deliminator[delim.length() + 1];
    strcpy(deliminator, delim.c_str());

    char *split_data;
    split_data = strtok(convert, deliminator);
    int count = 0;

    while (split_data != NULL)
    {
        // cout<<split_data<<endl;
        string char2string;
        char2string.append(split_data);
        // cout << char2string << endl;
        line_Data.push_back(char2string);
        if (count == 7)
        {
            break;
        }
        split_data = strtok(NULL, deliminator);
        count++;
    }
}

long int tajima::combos_N(int count)
{
    long int combinations;

    combinations = fact_half(count) / 2;

    return combinations;
}

long int tajima::fact_half(int count)
{
    long int tot = 1;
    for (int i = count; i > count - 2; i--)
    {
        // cout << tot;
        tot = tot * i;
    }
    return tot;
}

vector<string> tajima::compound_interpolationSearch(vector<pair<string, string>> &folder_Index, int &start_Co, int &end_Co)
{
    vector<string> file_List;

    vector<string> line_Data;
    split(line_Data, folder_Index[0].first, "_");
    int low_Value = stoi(line_Data[0]);
    split(line_Data, folder_Index[folder_Index.size() - 1].first, "_");
    int high_Value = stoi(line_Data[1]);
    // cout << "first: " << low_Value << " last: " << high_Value << endl;

    int start = 0;
    int end = folder_Index.size() - 1;

    while (start <= end && start_Co >= low_Value && start_Co <= high_Value)
    {
        vector<string> line_Data_get;

        int pos = start + ((double)(end - start) / ((high_Value - low_Value)) * (start_Co - low_Value));
        // cout << pos << endl;

        split(line_Data_get, folder_Index[pos].first, "_");
        int low_Value_atpos = stoi(line_Data_get[0]);
        int high_Value_atpos = stoi(line_Data_get[1]);

        if ((start_Co >= low_Value_atpos && start_Co <= high_Value_atpos) || (start_Co <= low_Value_atpos && end_Co <= high_Value_atpos) || (start_Co <= low_Value_atpos && end_Co >= high_Value_atpos))
        {
            // cout << low_Value_atpos << "_" << high_Value_atpos << endl;
            // backward_Search(pos, backward_get, folder_Index, start_Co, end_Co);

            // thread backward(&tajima::backward_Search, this, pos, ref(backward_get), ref(folder_Index), ref(start_Co), ref(end_Co));
            // thread forward(&tajima::forward_Search, this, pos, ref(forward_get), ref(folder_Index), ref(start_Co), ref(end_Co));

            // backward.join();
            // forward.join();

            vector<int> backward_get;
            vector<int> forward_get;

            future<vector<int>> backward_thread = async(&tajima::backward_Search, this, pos, folder_Index, start_Co, end_Co);
            future<vector<int>> forward_thread = async(&tajima::forward_Search, this, pos, folder_Index, start_Co, end_Co);

            backward_get = backward_thread.get();
            forward_get = forward_thread.get();

            for (auto positions : backward_get)
            {
                file_List.push_back(folder_Index[positions].second);
            }

            // cout << "caught :" << folder_Index[pos].second << endl;
            file_List.push_back(folder_Index[pos].second);

            for (auto positions : forward_get)
            {
                file_List.push_back(folder_Index[positions].second);
            }

            break;
        }
        else if (start_Co > low_Value_atpos)
        {
            start = pos + 1;
        }
        else
        {
            end = pos - 1;
        }

        split(line_Data_get, folder_Index[start].first, "_");
        low_Value = stoi(line_Data_get[0]);

        split(line_Data_get, folder_Index[end].first, "_");
        high_Value = stoi(line_Data_get[1]);
    }

    return file_List;
}

vector<int> tajima::backward_Search(int pos, vector<pair<string, string>> folder_Index, int start_Co, int end_Co)
{
    vector<int> backward_get;
    pos = pos - 1;
    vector<string> line_Data_get;
    while (pos >= 0)
    {
        split(line_Data_get, folder_Index[pos].first, "_");
        int low_Value_atpos = stoi(line_Data_get[0]);
        int high_Value_atpos = stoi(line_Data_get[1]);

        if (start_Co > high_Value_atpos)
        {
            break;
        }

        if ((start_Co >= low_Value_atpos && start_Co <= high_Value_atpos) || (start_Co <= low_Value_atpos && end_Co <= high_Value_atpos) || (start_Co <= low_Value_atpos && end_Co >= high_Value_atpos))
        {
            backward_get.push_back(pos);
        }

        pos = pos - 1;
    }
    return backward_get;
}

vector<int> tajima::forward_Search(int pos, vector<pair<string, string>> folder_Index, int start_Co, int end_Co)
{
    vector<int> forward_get;
    pos = pos + 1;
    vector<string> line_Data_get;
    while (pos < folder_Index.size())
    {
        split(line_Data_get, folder_Index[pos].first, "_");
        int low_Value_atpos = stoi(line_Data_get[0]);
        int high_Value_atpos = stoi(line_Data_get[1]);

        if (end_Co < low_Value_atpos)
        {
            break;
        }

        if ((start_Co >= low_Value_atpos && start_Co <= high_Value_atpos) || (start_Co <= low_Value_atpos && end_Co <= high_Value_atpos) || (start_Co <= low_Value_atpos && end_Co >= high_Value_atpos))
        {
            forward_get.push_back(pos);
        }

        pos = pos + 1;
    }
    return forward_get;
}

__global__ void a_Calculation(int N, float *a1_CUDA, float *a2_CUDA)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < N)
    {
        a1_CUDA[tid] = (float)1 / (tid + 1);
        a2_CUDA[tid] = (float)1 / ((tid + 1) * (tid + 1));
        tid += blockDim.x * gridDim.x;
    }
}

__global__ void add_Cuda(const float *a, float *out, int arraySize)
{
    int idx = threadIdx.x;
    float sum = 0;
    for (int i = idx; i < arraySize; i += 1024)
        sum += a[i];
    __shared__ int r[1024];
    r[idx] = sum;
    __syncthreads();
    for (int size = 1024 / 2; size > 0; size /= 2)
    { // uniform
        if (idx < size)
            r[idx] += r[idx + size];
        __syncthreads();
    }
    if (idx == 0)
        *out = r[0];
}

void tajima::calc_Pre(int &N_tot, float &a1, float &e1, float &e2)
{
    int N = N_tot - 1;
    float *a1_CUDA, *a2_CUDA;
    float *a1_partial, *a2_partial, a2;

    a1_partial = (float *)malloc(N * sizeof(float));
    a2_partial = (float *)malloc(N * sizeof(float));

    hipMallocManaged(&a1_CUDA, N * sizeof(int));
    hipMallocManaged(&a2_CUDA, N * sizeof(int));

    a_Calculation<<<tot_Blocks, tot_ThreadsperBlock>>>(N, a1_CUDA, a2_CUDA);
    hipDeviceSynchronize();

    hipMemcpy(a1_partial, a1_CUDA, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(a2_partial, a2_CUDA, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(a1_CUDA);
    hipFree(a2_CUDA);

    a1 = 0;
    a2 = 0;
    for (size_t i = 0; i < N; i++)
    {
        a1 += a1_partial[i];
        a2 += a2_partial[i];
    }

    cout << "a1\t: " << a1 << "\t"
         << "a2: " << a2 << endl;

    free(a1_partial);
    free(a2_partial);

    float b1 = (float)(N_tot + 1) / (3 * (N_tot - 1));
    float b2 = (float)(2 * (pow(N_tot, 2.0) + N_tot + 3)) / ((9 * N_tot) * (N_tot - 1));

    cout << "b1\t: " << b1 << "\t"
         << "b2: " << b2 << endl;

    float c1 = b1 - (1 / a1);
    float c2 = b2 - ((N_tot + 2) / (a1 * N_tot)) + (a2 / pow(a1, 2.0));

    cout << "c1\t: " << c1 << "\t"
         << "c2: " << c2 << endl;

    e1 = c1 / a1;
    e2 = c2 / (pow(a1, 2.0) + a2);

    cout << "e1\t: " << e1 << "\t"
         << "e2: " << e2 << endl;

    cout << endl;
}

int tajima::getN_Split(string file)
{
    fstream file_nCount;
    file_nCount.open(file);

    string header;
    getline(file_nCount, header);
    file_nCount.close();

    vector<string> header_Columns;
    split(header_Columns, header, "\t");

    int N = header_Columns.size() - 9;
    // cout << N << endl;
    return N;
}

vector<pair<string, string>> tajima::index_Folder(string &country)
{
    cout << "Initiating indexing folder\t: " << country << endl;
    string country_Only = country.substr(country.find_last_of("/") + 1, country.length());

    vector<string> index_pass_1;
    vector<pair<string, string>> file_coordinate;

    for (const auto &entry : filesystem::directory_iterator(country))
    {
        // cout << entry.path() << endl;
        string coordinates = entry.path().string();
        int trim_start = coordinates.find(country_Only + "_") + country_Only.length() + 1;
        int trim_end = coordinates.find_last_of(".") - trim_start;
        string trim = coordinates.substr(trim_start, trim_end);
        index_pass_1.push_back(trim);
        file_coordinate.push_back(make_pair(trim, coordinates));
    }

    vector<pair<int, int>> start_stop;
    vector<int> starts;

    for (string file : index_pass_1)
    {
        vector<string> file_start_end;
        split(file_start_end, file, "_");
        starts.push_back(stoi(file_start_end[0]));
        start_stop.push_back(make_pair(stoi(file_start_end[0]), stoi(file_start_end[1])));
    }

    sort(starts.begin(), starts.end());

    vector<pair<string, string>> sorted_Index;

    for (int nums : starts)
    {
        // cout << nums << endl;
        for (auto index_check : start_stop)
        {
            if (index_check.first == nums)
            {
                string sort_Line = to_string(index_check.first) + "_" + to_string(index_check.second);
                for (auto coordinates : file_coordinate)
                {
                    if (coordinates.first == sort_Line)
                    {
                        sorted_Index.push_back(make_pair(sort_Line, coordinates.second));
                        break;
                    }
                }
                break;
            }
        }
    }

    return sorted_Index;
}

vector<string> tajima::get_Countries()
{
    vector<string> folders;
    for (auto &check : std::filesystem::recursive_directory_iterator(this->input_Folder))
    {
        if (check.is_directory())
        {
            folders.push_back(check.path().string());
        }
    }
    return folders;
}

void tajima::split(vector<string> &line_Data, string line, string delim)
{
    vector<string>().swap(line_Data);
    char *convert;
    string capture(line);
    convert = &capture[0];
    // cout<<convert;

    char deliminator[delim.length() + 1];
    strcpy(deliminator, delim.c_str());

    char *split_data;
    split_data = strtok(convert, deliminator);

    while (split_data != NULL)
    {
        // cout<<split_data<<endl;
        string char2string;
        char2string.append(split_data);
        // cout << char2string << endl;
        line_Data.push_back(char2string);
        split_data = strtok(NULL, deliminator);
    }

    // delete convert;
    // delete split_data;
}

void tajima::split_Convert(int *line_temp, string line, string delim)
{
    char *convert;
    string capture(line);
    convert = &capture[0];
    // cout<<convert;

    char deliminator[delim.length() + 1];
    strcpy(deliminator, delim.c_str());

    char *split_data;
    split_data = strtok(convert, deliminator);
    int count = 0;

    while (split_data != NULL)
    {
        // cout<<split_data<<endl;
        string char2string;
        char2string.append(split_data);
        // cout << char2string << "\t";
        // cout << char2string << endl;
        // line_Data.push_back(char2string);
        if (count >= 9)
        {
            // cout << "count : " << char2string << endl;
            if (char2string.substr(0, 3) == "0|0")
            {
                // cout << "0|0" << count - 9 << endl;
                line_temp[count - 9] = 0;
            }
            else if (char2string.substr(0, 3) == "0|1")
            {
                // cout << "0|1" << count - 9 << endl;
                line_temp[count - 9] = 1;
            }
            else if (char2string.substr(0, 3) == "1|0")
            {
                line_temp[count - 9] = 2;
            }
            else if (char2string.substr(0, 3) == "1|1")
            {
                line_temp[count - 9] = 3;
            }
            // cout << "count : " << count - 9 << "\t" << line_temp[count - 9] << endl;
        }
        count++;
        split_data = strtok(NULL, deliminator);
    }
    // cout << endl;
    // delete convert;
    // delete split_data;
}