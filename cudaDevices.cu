#include "cudaDevices.cuh"

cudaDevices::cudaDevices()
{

    cout << "Listing all CUDA capable devices:" << endl;
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++)
    {
        cout << endl;
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        cout << "GPU number\t: " << i << endl;
        cout << "GPU name\t: " << prop.name << endl;
        size_t l_free = 0;
        size_t l_Total = 0;
        hipError_t error_id = hipMemGetInfo(&l_free, &l_Total);
        cout << "GPU memory (GB)\t: " << l_Total / (1000 * 1000 * 1000) << endl;
        hipError_t err = hipGetLastError();

        if (err != hipSuccess)
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));

            // Possibly: exit(-1) if program cannot continue....
        }
        cout << "GPU number of multiprocessor(s)\t: " << prop.multiProcessorCount << endl;
        cout << "GPU block(s) per multiprocessor\t: " << prop.maxBlocksPerMultiProcessor << endl;
        cout << "GPU thread(s) per block\t: " << prop.maxThreadsPerBlock << endl;
        cout << endl;
    }
}