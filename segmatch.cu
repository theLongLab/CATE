#include "hip/hip_runtime.h"
#include "segmatch.cuh"

segmatch::segmatch(string parameter_Master_Location)
{
    cout << "Intializing seg matching\n";

   // exit(-1);

    parameter_load Parameters = parameter_load();
    functions_library function = functions_library();

    vector<string> parameters_List = {
        "\"Intermediate folders\"",
        "\"Output folders\"",
        "\"Nodes master profile\"",
        "\"Sequence master profile\"",
        "\"CPU cores\"",
        "\"Segregating match sequences\"",
        "\"Segregating match Node ID\"",
        "\"Segregating match tissue\"",
        "\"Segregating match cutoff\"",
        "\"Multi read\""};

    vector<string> found_Parameters = Parameters.get_parameters(parameter_Master_Location, parameters_List);

    intermediate_Folder_location = Parameters.get_STRING(found_Parameters[0]);
    output_Folder_location = Parameters.get_STRING(found_Parameters[1]);
    CPU_cores = Parameters.get_INT(found_Parameters[4]);

    if (function.to_Upper_Case(found_Parameters[9]) == "NO")
    {
        CPU_cores = 1;
    }

    cout << "\nConfiguring segregating sites: \n";
    string sequence_Master_location = Parameters.get_STRING(found_Parameters[3]);

    vector<pair<string, string>> mutations_Block = Parameters.get_block_from_File(sequence_Master_location, "Mutations");
    int num_mutation_Hotspots = Parameters.get_INT(mutations_Block, "Number of hotspots");

    if (num_mutation_Hotspots > 0)
    {
        cout << "\nProcessing " << num_mutation_Hotspots << " mutation hotspots: \n";
        vector<string> split_Region;
        for (int hotspot = 0; hotspot < num_mutation_Hotspots; hotspot++)
        {
            string hotspot_ID = "Hotspot " + to_string(hotspot + 1);
            cout << "\nProcessing: " << hotspot_ID << endl;
            vector<pair<string, string>> mutations_hotspot_Block = Parameters.get_block_from_block(mutations_Block, hotspot_ID);

            string region = Parameters.get_STRING(mutations_hotspot_Block, "Region");
            function.split(split_Region, region, '_');

            int start_Pos = stoi(split_Region[0]) - 1;
            int stop_Pos = stoi(split_Region[1]) - 1;
            positions_start_end.push_back(make_pair(start_Pos, stop_Pos));

            cout << "Start position: " << (start_Pos + 1) << " | Stop position: " << (stop_Pos + 1) << endl;
            int bases = stop_Pos - start_Pos + 1;
            cout << "Number of bases: " << bases << endl;
            total_Bases = total_Bases + bases;
        }

        cout << "\nTotal number of segregating sites to check: " << total_Bases << endl;

        tissue_Name = Parameters.get_STRING(found_Parameters[7]);
        cout << "\nIdentifying tissue index: " << tissue_Name << endl;

        string node_Master_location = Parameters.get_STRING(found_Parameters[2]);
        vector<pair<string, string>> Tissues_Block = Parameters.get_block_from_File(node_Master_location, "Tissue profiles");
        int num_Tissues = Parameters.get_INT(Tissues_Block, "Number of tissues");

        if (num_Tissues > 0)
        {
            for (int tissue = 0; tissue < num_Tissues; tissue++)
            {
                string check = "Tissue " + to_string(tissue + 1) + " Name";
                string Name = Parameters.get_STRING(Tissues_Block, check);

                if (Name == tissue_Name)
                {
                    tissue_Index = tissue;
                    break;
                }
            }

            if (tissue_Index != -1)
            {
                cout << "Index: " << tissue_Index << endl;

                cout << "\nGetting node index\n";
                string index_Node_File = intermediate_Folder_location + "/index_Data/node_Index.csv";
                node_ID = Parameters.get_STRING(found_Parameters[6]);
                cout << "Node: " << node_ID << endl;

                fstream index_File;
                index_File.open(index_Node_File, ios::in);

                if (index_File.is_open())
                {
                    string line;
                    getline(index_File, line);

                    vector<string> line_Data;

                    while (getline(index_File, line))
                    {
                        function.split(line_Data, line, '\t');
                        if (line_Data[1] == node_ID)
                        {
                            node_Index = stoi(line_Data[0]);
                            break;
                        }
                    }

                    index_File.close();

                    if (node_Index != -1)
                    {
                        cout << "Node index: " << node_Index << endl;
                        node_intermediary_location = intermediate_Folder_location + "/sequence_Data/" + to_string(node_Index) + "/" + to_string(tissue_Index);
                        node_results_Location = output_Folder_location + "/node_Data/" + node_ID;

                        cout << "\nProcessing query sequences\n";
                        cutoff = Parameters.get_FLOAT(found_Parameters[8]);
                        cout << "cutoff Percentage: " << cutoff << endl;
                        cutoff_Count = total_Bases * cutoff;
                        cout << "cutoff Count: " << cutoff_Count << endl;

                        string query_Sequence_Location = Parameters.get_STRING(found_Parameters[5]);
                        cout << "Reading sequences file: " << query_Sequence_Location << endl;

                        fstream sequences_File;
                        sequences_File.open(query_Sequence_Location, ios::in);

                        if (sequences_File.is_open())
                        {
                            string line;
                            while (getline(sequences_File, line))
                            {
                                if (line.at(0) == '>')
                                {
                                    string line_Fix = function.clean_Invisible(line.substr(1, line.length()));
                                    sequences_to_Check_ID.push_back(function.clean_Line(line_Fix));
                                }
                                else
                                {
                                    string n_Line = line;
                                    for (int base = 0; base < line.size(); base++)
                                    {
                                        if (n_Line.at(base) == 'A' || n_Line.at(base) == 'a')
                                        {
                                            n_Line.at(base) = '0';
                                        }
                                        else if (n_Line.at(base) == 'T' || n_Line.at(base) == 't')
                                        {
                                            n_Line.at(base) = '1';
                                        }
                                        else if (n_Line.at(base) == 'G' || n_Line.at(base) == 'g')
                                        {
                                            n_Line.at(base) = '2';
                                        }
                                        else if (n_Line.at(base) == 'C' || n_Line.at(base) == 'c')
                                        {
                                            n_Line.at(base) = '3';
                                        }
                                    }

                                    sequences_to_Check.push_back(n_Line);
                                }
                            }
                            sequences_File.close();

                            if (sequences_to_Check.size() == sequences_to_Check_ID.size())
                            {
                                cout << "Sequences to check: " << sequences_to_Check.size() << endl;
                                cout << "\nMulti read: " << Multi_Read << endl;
                                cout << "CPU cores being used: " << CPU_cores << endl;

                                cout << "\nConfiguring output folders: " << endl;
                                function.config_Folder(node_results_Location + "/seg_Match", "Master output");

                                for (int sequence = 0; sequence < sequences_to_Check.size(); sequence++)
                                {
                                    function.config_Folder(node_results_Location + "/seg_Match/" + sequences_to_Check_ID[sequence], sequences_to_Check_ID[sequence]);
                                    function.create_File(node_results_Location + "/seg_Match/" + sequences_to_Check_ID[sequence] + "/" + tissue_Name + "_" + to_string(cutoff) + ".csv", "Target_sequence\tTissue\tGeneration\tquery_Sequence_ID\tmatching_Percentage\tmatch_Count\tmismatch_Count\tMismatch_bases");
                                }
                            }
                            else
                            {
                                cout << "IDs and sequences do not match in count:" << sequences_to_Check_ID.size() << "\t" << sequences_to_Check.size() << endl;
                                exit(-1);
                            }
                        }
                        else
                        {
                            cout << "UNABLE TO OPEN QUERY SEQUENCE FILE: " << query_Sequence_Location << endl;
                            exit(-1);
                        }
                    }
                    else
                    {
                        cout << "UNABLE TO FIND NODE: " << node_ID << endl;
                        exit(-1);
                    }
                }
                else
                {
                    cout << "UNABLE TO OPEN NODE INDEX FILE: " << index_Node_File << endl;
                    exit(-1);
                }
            }
            else
            {
                cout << "TISSUE " << tissue_Name << " NOT FOUND\n";
                exit(-1);
            }
        }
        else
        {
            cout << "HAS TO HAVE AT LEAST 1 TISSUE\n";
            exit(-1);
        }
    }
    else
    {
        cout << "HAS TO HAVE AT LEAST 1 MUTATION HOTSPOT REGION\n";
        exit(-1);
    }
}

void segmatch::ingress()
{
    functions_library functions = functions_library();
    vector<string> line_Data;

    cout << "\nIntializing generation detection\n";
    vector<pair<int, string>> generations_Paths;

    for (const auto &entry : filesystem::directory_iterator(node_intermediary_location))
    {
        if (filesystem::is_directory(entry))
        {
            // cout << entry.path().string() << endl;
            string gen_Name = entry.path().filename().string();
            functions.split(line_Data, gen_Name, '_');
            generations_Paths.push_back(make_pair(stoi(line_Data[1]), entry.path().string()));
        }
    }

    sort(generations_Paths.begin(), generations_Paths.end());

    int num_per_Core = sequences_to_Check_ID.size() / CPU_cores;
    int remainder = sequences_to_Check_ID.size() % CPU_cores;

    for (int generation = 0; generation < generations_Paths.size(); generation++)
    {
        cout << "\nProcessing generation " << generation + 1 << " of " << generations_Paths.size() << endl;
        cout << "Current generation: " << generations_Paths[generation].first << endl;

        vector<pair<int, int>> nFASTA_files = functions.index_Source_folder(generations_Paths[generation].second);

        cout << "Detecting haplotypes and their counts: \n";

        for (int nFASTA_file = 0; nFASTA_file < nFASTA_files.size(); nFASTA_file++)
        {
            cout << "Processing sequence file " << nFASTA_file + 1 << " of " << nFASTA_files.size() << endl;

            string n_FASTA_location = generations_Paths[generation].second + "/" + to_string(nFASTA_files[nFASTA_file].first) + "_" + to_string(nFASTA_files[nFASTA_file].second) + ".nfasta";
            cout << "Reading file: " << n_FASTA_location << endl;

            fstream nFASTA;
            nFASTA.open(n_FASTA_location, ios::in);

            vector<string> headers;
            vector<string> lines;

            if (nFASTA.is_open())
            {
                string line;
                string header = "";
                while (getline(nFASTA, line))
                {
                    if (line.at(0) == '>')
                    {
                        string header = functions.clean_Invisible(line.substr(1, line.length()));
                        header = functions.clean_Line(header);
                        functions.split(line_Data, header, '_');
                        headers.push_back(functions.clean_Line(line_Data[0]));
                    }
                    else
                    {
                        lines.push_back(line);
                    }
                }
                nFASTA.close();
            }
            else
            {
                cout << "UNABLE TO OPEN nFASTA FILE: " << n_FASTA_location << endl;
                exit(-1);
            }

            cout << "Processing file:" << endl;

            for (int line_Num = 0; line_Num < lines.size(); line_Num++)
            {
                string line = lines[line_Num];
                vector<thread> threads_vec;

                for (int core_ID = 0; core_ID < CPU_cores; core_ID++)
                {
                    int start_Node = core_ID * num_per_Core;
                    int stop_Node = start_Node + num_per_Core;

                    threads_vec.push_back(thread{&segmatch::get_Match, this, start_Node, stop_Node, ref(line), headers[line_Num], to_string(generations_Paths[generation].first)});
                }

                if (remainder != 0)
                {
                    int start_Node = sequences_to_Check_ID.size() - remainder;
                    int stop_Node = sequences_to_Check_ID.size();

                    threads_vec.push_back(thread{&segmatch::get_Match, this, start_Node, stop_Node, ref(line), headers[line_Num], to_string(generations_Paths[generation].first)});
                }

                for (thread &t : threads_vec)
                {
                    if (t.joinable())
                    {
                        t.join();
                    }
                }

                threads_vec.clear();
            }
            // exit(-1);
        }
    }
    cout << "\nDone\n";
}

void segmatch::get_Match(int start, int stop, string &sequence_Query, string header, string generation)
{
    for (int sequence = start; sequence < stop; sequence++)
    {
        int match_Count = 0;
        vector<pair<int, string>> mismatch_Bases;

        for (int site = 0; site < positions_start_end.size(); site++)
        {
            for (int base = positions_start_end[site].first; base <= positions_start_end[site].second; base++)
            {
                // cout << sequences_to_Check[sequence].at(base) << "\t" << sequence_Query.at(base) << endl;
                if (sequences_to_Check[sequence].at(base) == sequence_Query.at(base))
                {
                    match_Count++;
                }
                else
                {
                    // cout << sequences_to_Check[sequence].at(base) << "\t" << sequence_Query.at(base) << "\t" << to_string(sequence_Query.at(base)) << endl;
                    if (sequence_Query.at(base) == '0')
                    {
                        mismatch_Bases.push_back(make_pair(base, "A"));
                    }
                    else if (sequence_Query.at(base) == '1')
                    {
                        mismatch_Bases.push_back(make_pair(base, "T"));
                    }
                    else if (sequence_Query.at(base) == '2')
                    {
                        mismatch_Bases.push_back(make_pair(base, "G"));
                    }
                    else if (sequence_Query.at(base) == '3')
                    {
                        mismatch_Bases.push_back(make_pair(base, "C"));
                    }
                }
            }
        }

        float match_Percentage = (float)match_Count / (float)total_Bases;

        if (match_Percentage >= cutoff)
        {
            fstream write_File;
            write_File.open(node_results_Location + "/seg_Match/" + sequences_to_Check_ID[sequence] + "/" + tissue_Name + "_" + to_string(cutoff) + ".csv", ios::app);
            cout << "Writing to file: " << node_results_Location + "/seg_Match/" + sequences_to_Check_ID[sequence] + "/" + tissue_Name + "_" + to_string(cutoff) + ".csv" << endl;

            if (write_File.is_open())
            {
                write_File << sequences_to_Check_ID[sequence] << "\t" << tissue_Name << "\t" << generation << "\t"
                           << node_ID << "_" << tissue_Name << "_" << generation << "_" << header << "\t" << match_Percentage << "\t" << match_Count << "\t" << to_string(mismatch_Bases.size()) << "\t";
                if (mismatch_Bases.size() > 0)
                {
                    for (int mismatch = 0; mismatch < mismatch_Bases.size(); mismatch++)
                    {
                        if (mismatch != 0)
                        {
                            write_File << "|";
                        }
                        write_File << to_string(mismatch_Bases[mismatch].first + 1) << ":" << mismatch_Bases[mismatch].second;
                    }
                }
                else
                {
                    write_File << "NA";
                }

                write_File << endl;

                write_File.close();
            }
            // cout << sequences_to_Check_ID[sequence] << endl;
            // cout << match_Count << "/" << total_Bases << endl;
            // cout << match_Percentage << endl;
        }
        // else
        // {
        //     cout << match_Count << endl;
        // }
    }
}